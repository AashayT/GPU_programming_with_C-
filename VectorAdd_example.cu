
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(long int *a, long int *b, long int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2*2048*2048*2)
#define THREADS_PER_BLOCK 512

int main()
{
    	long int *a, *b, *c;
	long int *d_a, *d_b, *d_c;
	long int size = N * sizeof(long int );
	
	printf("Value of N=%d \n", N);
	/* allocate space for device copies of a, b, c */
	
	printf("Memory allocation for GPU device\n");
	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */
	printf("Memory allocation for CPU\n");
	a = (long int *)malloc( size );
	b = (long int *)malloc( size );
	c = (long int *)malloc( size );
	
	printf("Defining the numbers\n");
	for( long int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	vector_add<<<(N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %d\n",c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
